#include "head.h"

float *h_a;
float *h_b;

float *d_a;

void CPU_malloc(){

	size_t size = N*sizeof(float);

	h_a = (float *)malloc(size);
	h_b = (float *)malloc(size);
}

void GPU_malloc(){

	size_t size = N*sizeof(float);

	hipError_t Error;

	Error = hipMalloc((void**)&d_a,size);
	printf("CUDA error(malloc d_a) = %s\n", hipGetErrorString(Error));
}

void Free(){

	free(h_a);
	free(h_b);

	hipFree(d_a);
}

void Init(){

	int i;
	for(i=0;i<N;i++){
		h_a[i] = i;
	}
}

void print(float *a){

	int i;
	for(i=0;i<N;i++){
		printf("%f ", a[i]);
	}
	printf("\n");

}

void Sent_to_device(){

	size_t size = N*sizeof(float);
	hipError_t Error;

	Error = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	printf("CUDA error(copy h_a) = %s\n", hipGetErrorString(Error));
}

void Sent_to_host(){

        size_t size = N*sizeof(float);
        hipError_t Error;

        Error = hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost);
	printf("CUDA error(copy d_a) = %s\n", hipGetErrorString(Error));
}

