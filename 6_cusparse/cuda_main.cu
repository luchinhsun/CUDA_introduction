#include "head.h"

//variable for cusparse
hipsparseStatus_t status;
hipsparseHandle_t handle=0;
hipsparseMatDescr_t descr=0;
hipsparseMatDescr_t descrL=0;
hipsparseMatDescr_t descrU=0;
int *cooRowIndexHostPtr;
int * cooColIndexHostPtr;
float * cooValHostPtr;
int *cooRowIndex;
int * cooColIndex;
float * cooVal;
float * cooValLU;
float * yHostPtr;
float * y;
float * xHostPtr;
float * x;
float * temp;
int * csrRowPtr;

float dzero =0.0;
float done =1.0;
float dtwo =2.0;
float dthree=3.0;
float dfive =5.0;

void Allocate_Memory(){
	//cusparse
	size_t size = nnz*sizeof(int);
	cooRowIndexHostPtr = (int *) malloc(size);
	cooColIndexHostPtr = (int *) malloc(size);
	cooValHostPtr = (float *)malloc(nnz*sizeof(float));
	cooRowIndexHostPtr[0]=0; cooColIndexHostPtr[0]=0; cooValHostPtr[0]=1.0;
	cooRowIndexHostPtr[1]=0; cooColIndexHostPtr[1]=2; cooValHostPtr[1]=2.0;
	cooRowIndexHostPtr[2]=0; cooColIndexHostPtr[2]=3; cooValHostPtr[2]=3.0;
	cooRowIndexHostPtr[3]=1; cooColIndexHostPtr[3]=1; cooValHostPtr[3]=4.0;
	cooRowIndexHostPtr[4]=2; cooColIndexHostPtr[4]=0; cooValHostPtr[4]=5.0;
	cooRowIndexHostPtr[5]=2; cooColIndexHostPtr[5]=2; cooValHostPtr[5]=6.0;
	cooRowIndexHostPtr[6]=2; cooColIndexHostPtr[6]=3; cooValHostPtr[6]=7.0;
	cooRowIndexHostPtr[7]=3; cooColIndexHostPtr[7]=1; cooValHostPtr[7]=8.0;
	cooRowIndexHostPtr[8]=3; cooColIndexHostPtr[8]=3; cooValHostPtr[8]=9.0;

	yHostPtr    = (float *)malloc(n*sizeof(float));
	yHostPtr[0] = 50.0; yHostPtr[1] = 60.0; yHostPtr[2] = 70.0; yHostPtr[3] = 80.0;
	xHostPtr    = (float *)malloc(n*sizeof(float));
	xHostPtr[0] = 0.0; xHostPtr[1] = 0.0; xHostPtr[2] = 0.0; xHostPtr[3] = 0.0;

	hipError_t Error;

	Error = hipMalloc((void**)&cooRowIndex, size);
	printf("CUDA error(malloc RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooColIndex, size);
	printf("CUDA error(malloc ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooVal, nnz*sizeof(float));
	printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooValLU, nnz*sizeof(float));
        printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&y, n*sizeof(float));
	printf("CUDA error(malloc y) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&x, n*sizeof(float));
        printf("CUDA error(malloc x) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&temp, n*sizeof(float));
        printf("CUDA error(malloc temp) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&csrRowPtr,(n+1)*sizeof(int));
        printf("CUDA error(malloc csrRowPtr) = %s\n",hipGetErrorString(Error));

	status= hipsparseCreate(&handle);
	status= hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	status = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL,HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
    	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

    	status = hipsparseCreateMatDescr(&descrU);
    	hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
    	hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
    	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
    	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
}

void Send_To_Device(){
	hipError_t Error;
	size_t size = nnz*sizeof(int);
	Error = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooColIndex, cooColIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy Val) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(y, yHostPtr, (size_t)(n*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy y) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(x, xHostPtr, (size_t)(n*sizeof(float)), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy x) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	status= hipsparseXcoo2csr(handle,cooRowIndex,nnz,n, csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("shit");
	}

	status= hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz,
				&dtwo, descr, cooVal, csrRowPtr, cooColIndex, &y[0], &dzero, &y[0]);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("shit");
	}

	cusparseSolveAnalysisInfo_t infoA = 0;
        status = cusparseCreateSolveAnalysisInfo(&infoA);

        cusparseSolveAnalysisInfo_t info_u = 0;
        status = cusparseCreateSolveAnalysisInfo(&info_u);

	status= cusparseScsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, descr,
						cooVal, csrRowPtr, cooColIndex, infoA);
	hipMemcpy(cooValLU, cooVal, nnz*sizeof(float), hipMemcpyDeviceToDevice);
	status = cusparseScsrilu0(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, descr,
						cooValLU, csrRowPtr, cooColIndex, infoA);
	status = cusparseScsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, descrU,
						cooVal, csrRowPtr, cooColIndex, info_u);
	status = cusparseScsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, &done, descrL,
                                              cooValLU, csrRowPtr, cooColIndex, infoA, &y[0], &temp[0]);
	status = cusparseScsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, &done, descrU,
                                              cooValLU, csrRowPtr, cooColIndex, info_u, &temp[0] , &x[0]);

	if (status != HIPSPARSE_STATUS_SUCCESS) {
                printf("shit");
        }

	cusparseDestroySolveAnalysisInfo(infoA);
        cusparseDestroySolveAnalysisInfo(info_u);

}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(yHostPtr, y, (size_t)(n*sizeof(float)), hipMemcpyDeviceToHost);
	printf("CUDA error(memcpy y->yHostPtr) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(xHostPtr, x, (size_t)(n*sizeof(float)), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy x->xHostPtr) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){

	status = hipsparseDestroyMatDescr(descr); descr = 0; 
	status = hipsparseDestroy(handle); handle = 0;

	if (yHostPtr) free(yHostPtr);
	if (xHostPtr) free(xHostPtr);
        if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);
        if (cooColIndexHostPtr) free(cooColIndexHostPtr);
        if (cooValHostPtr) free(cooValHostPtr);
        if (y) hipFree(y);
	if (x) hipFree(x);
	if (temp) hipFree(temp);
        if (csrRowPtr) hipFree(csrRowPtr);
        if (cooRowIndex) hipFree(cooRowIndex);
        if (cooColIndex) hipFree(cooColIndex);
        if (cooVal) hipFree(cooVal);
	if (cooValLU) hipFree(cooValLU);
        if (descr) hipsparseDestroyMatDescr(descr);
        if (handle) hipsparseDestroy(handle);

}
