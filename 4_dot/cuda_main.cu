#include "hip/hip_runtime.h"
#include "head.h"

float *h_a;
float *h_b;
float *h_c;

float *d_a;
float *d_b;
float *d_c;

float *d_temp;

void CPU_malloc(){

	size_t size = N*sizeof(float);

	h_a = (float *)malloc(size);
	h_b = (float *)malloc(size);
	h_c = (float *)malloc(sizeof(float));
}

void GPU_malloc(){

	size_t size = N*sizeof(float);

	hipError_t Error;

	Error = hipMalloc((void**)&d_a,size);
	printf("CUDA error(malloc d_a) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_b,size);
        printf("CUDA error(malloc d_b) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_c,sizeof(float));
        printf("CUDA error(malloc d_c) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_temp,(N+256-1)/256*sizeof(float));
        printf("CUDA error(malloc d_temp) = %s\n", hipGetErrorString(Error));

}

void Free(){

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_temp);
}

void Init(){

	int i;
	for(i=0;i<N;i++){
		h_a[i] = i;
		h_b[i] = 2*i;
	}
}

void print(float *a){

	int i;
	for(i=0;i<N;i++){
		printf("%f ", a[i]);
	}
	printf("\n");

}

void Sent_to_device(){

	size_t size = N*sizeof(float);
	hipError_t Error;

	Error = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	printf("CUDA error(copy h_a) = %s\n", hipGetErrorString(Error));
	Error = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
        printf("CUDA error(copy h_b) = %s\n", hipGetErrorString(Error));
}

void Sent_to_host(){

        size_t size = sizeof(float);
        hipError_t Error;

        Error = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	printf("CUDA error(copy d_c) = %s\n", hipGetErrorString(Error));
}

void product(){
	int i;

	for(i=0;i<N;i++){
                h_c[0] += h_a[i]*h_b[i];
        }
}

__global__ void GPU_DotProduct(float *x, float *y, float *z, int n){
        int I = threadIdx.x;
        int i = blockDim.x * blockIdx.x + I;

        __shared__ float temp[256];
        temp[I] = 0.0;
        if(i<n){
                temp[I] = x[i] * y[i];
        }

        __syncthreads();

        for (int stride = blockDim.x/2; stride>0; stride = stride/2){
                if(I<stride){
                        temp[I] = temp[I] + temp[I+stride];
                }
                __syncthreads();
        }

        if(I==0){
                z[blockIdx.x] = temp[0];
        }

}

__global__ void GPU_DotProduct_Sum(float *x, float *y, int y_location, int n){

        int i;
        float sum = 0.0;
        for(i=0; i<n; i++){
                sum += x[i];
        }
        y[y_location] = sum;
}

__global__ void GPU_DotProduct_Sum_reduction(float *x, float *y, int y_location){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	for (int stride = blockDim.x/2; stride>0; stride = stride/2){
		if(i<stride){
			x[i] = x[i] + x[i+stride];
		}
		__syncthreads();
	}

	if(i==0){
		y[y_location] = x[0];
	}
}

void DotFunction(){
        int tpb = 256;
        int bpg = (N+tpb-1)/tpb;

        GPU_DotProduct<<<bpg, tpb>>>(d_a, d_b, d_temp, N);
        //GPU_DotProduct_Sum<<<1, 1>>>(d_temp, d_c, 0, bpg);
	GPU_DotProduct_Sum_reduction<<<1, bpg>>>(d_temp, d_c, 0);
}
