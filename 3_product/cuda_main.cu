#include "hip/hip_runtime.h"
#include "head.h"

float *h_a;
float *h_b;
float *h_c;

float *d_a;
float *d_b;
float *d_c;

void CPU_malloc(){

	size_t size = N*sizeof(float);

	h_a = (float *)malloc(size);
	h_b = (float *)malloc(size);
	h_c = (float *)malloc(size);
}

void GPU_malloc(){

	size_t size = N*sizeof(float);

	hipError_t Error;

	Error = hipMalloc((void**)&d_a,size);
	printf("CUDA error(malloc d_a) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_b,size);
        printf("CUDA error(malloc d_b) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_c,size);
        printf("CUDA error(malloc d_c) = %s\n", hipGetErrorString(Error));
}

void Free(){

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void Init(){

	int i;
	for(i=0;i<N;i++){
		h_a[i] = i;
		h_b[i] = 2*i;
	}
}

void print(float *a){

	int i;
	for(i=0;i<N;i++){
		printf("%f ", a[i]);
	}
	printf("\n");

}

void Sent_to_device(){

	size_t size = N*sizeof(float);
	hipError_t Error;

	Error = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	printf("CUDA error(copy h_a) = %s\n", hipGetErrorString(Error));
	Error = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
        printf("CUDA error(copy h_b) = %s\n", hipGetErrorString(Error));
}

void Sent_to_host(){

        size_t size = N*sizeof(float);
        hipError_t Error;

        Error = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	printf("CUDA error(copy d_c) = %s\n", hipGetErrorString(Error));
}

__global__ void GPU_product(float *d_a, float *d_b, float *d_c){
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if(i<N)	d_c[i] = d_a[i]*sqrt(d_b[i]);
}

__global__ void GPU_product_serial(float *d_a, float *d_b, float *d_c){
        int i = blockDim.x*blockIdx.x + threadIdx.x;

        for(i=0;i<N;i++){
		d_c[i] = d_a[i]*sqrt(d_b[i]);
	}
}

void product1(){
	int i;

	for(i=0;i<N;i++){
                h_c[i] = h_a[i]*sqrt(h_b[i]);
        }
}

void product2(){
        int bpg = 1;
        int tpb = 1;

        GPU_product_serial<<<bpg, tpb>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
}

void product3(){
	int bpg = 256;
	int tpb = (N+bpg-1)/bpg;

	GPU_product<<<bpg, tpb>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
}

