#include "hip/hip_runtime.h"
#include "head.h"

float *h_a;
float *h_b;
float *h_c;

float *d_a;
float *d_b;
float *d_c;

void CPU_malloc(){

	size_t size = N*sizeof(float);

	h_a = (float *)malloc(N*size);
	h_b = (float *)malloc(size);
	h_c = (float *)malloc(size);
}

void GPU_malloc(){

	size_t size = N*sizeof(float);

	hipError_t Error;

	Error = hipMalloc((void**)&d_a,N*size);
	printf("CUDA error(malloc d_a) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_b,size);
        printf("CUDA error(malloc d_b) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_c,size);
        printf("CUDA error(malloc d_c) = %s\n", hipGetErrorString(Error));
}

void Free(){

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void Init(){

	int i;
	for(i=0;i<N*N;i++){
		h_a[i] = i;
	}
	for(i=0;i<N;i++){
		h_b[i] = 2*i;
	}
}

void print(float *a){

	int i;
	for(i=0;i<N;i++){
		printf("%f ", a[i]);
	}
	printf("\n");

}

void print_matrix(float *a){

        int i, j;
        for(i=0;i<N;i++){
		for(j=0;j<N;j++){
                	printf("%f ", a[i*N+j]);
		}
        	printf("\n");
	}

}


void Sent_to_device(){

	size_t size = N*sizeof(float);
	hipError_t Error;

	Error = hipMemcpy(d_a, h_a, N*size, hipMemcpyHostToDevice);
	printf("CUDA error(copy h_a) = %s\n", hipGetErrorString(Error));
	Error = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
        printf("CUDA error(copy h_b) = %s\n", hipGetErrorString(Error));
}

void Sent_to_host(){

        size_t size = N*sizeof(float);
        hipError_t Error;

        Error = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	printf("CUDA error(copy d_c) = %s\n", hipGetErrorString(Error));
}

__global__ void GPU_product(float *x, float *y, float *z){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j;

	if(i<N){
		z[i] = 0.0;
		for(j=0;j<N;j++){
			z[i] += x[i*N+j]*y[j];
		}
	}
}

void product1(){
	int i, j;

	for(i=0;i<N;i++){
		h_c[i] = 0.0;
		for(j=0;j<N;j++){
                	h_c[i] += h_a[i*N+j]*h_b[j];
		}
        }
}

void product2(){
	int tpb = 256;
	int bpg = (N+tpb-1)/tpb;

	GPU_product<<<bpg, tpb>>>(d_a, d_b, d_c);
}

