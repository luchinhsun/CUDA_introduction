#include "hip/hip_runtime.h"
#include "head.h"

float *h_a;
float *h_b;
float *h_c;

float *d_a;
float *d_b;
float *d_c;

void CPU_malloc(){

	size_t size = N*sizeof(float);

	h_a = (float *)malloc(size);
	h_b = (float *)malloc(size);
	h_c = (float *)malloc(size);
}

void GPU_malloc(){

	size_t size = N*sizeof(float);

	hipError_t Error;

	Error = hipMalloc((void**)&d_a,size);
	printf("CUDA error(malloc d_a) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_b,size);
        printf("CUDA error(malloc d_b) = %s\n", hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_c,size);
        printf("CUDA error(malloc d_c) = %s\n", hipGetErrorString(Error));
}

void Free(){

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void Init(){

	int i;
	for(i=0;i<N;i++){
		h_a[i] = i;
		h_b[i] = 2*i;
	}
}

void print(float *a){

	int i, j;
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%f ", a[i*n+j]);
		}
		printf("\n");
	}

}

void Sent_to_device(){

	size_t size = N*sizeof(float);
	hipError_t Error;

	Error = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	printf("CUDA error(copy h_a) = %s\n", hipGetErrorString(Error));
	Error = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
        printf("CUDA error(copy h_b) = %s\n", hipGetErrorString(Error));
}

void Sent_to_host(){

        size_t size = N*sizeof(float);
        hipError_t Error;

        Error = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	printf("CUDA error(copy d_c) = %s\n", hipGetErrorString(Error));
}

__global__ void GPU_product(float *d_a, float *d_b, float *d_c){
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	if(i<N)	d_c[i] = d_a[i]+d_b[i];
}

void CPU_add(){
	int i, j;

	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
                	h_c[i*n+j] = h_a[i*n+j]+h_b[i*n+j];
		}
        }
}

void GPU_add(){
	int tpb = 256;
	int bpg = (N+tpb-1)/tpb;

	GPU_product<<<bpg, tpb>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
}

