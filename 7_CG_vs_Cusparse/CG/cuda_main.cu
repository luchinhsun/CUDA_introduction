#include "hip/hip_runtime.h"
#include "head.h"

// Define our variables of interest
float *h_A;     // A matrix
float *h_B;     // B vector
float *h_x;     // x (solution) vector
float *h_R;     // Residual vector
float *h_P;
float *h_AP;
float *h_scalars;  // Scalars

// Define GPU Variables
float *d_A;
float *d_B;
float *d_x;
float *d_R;
float *d_P;
float *d_AP;
float *d_scalars;
float *d_temp;

clock_t start_cu;
clock_t end_cu;
float time_used_cu;

float gpu_timer;

void Allocate_Memory() {

        size_t size;

        // Our N*N variable (A)
        size = N*N*sizeof(float);
        h_A = (float*)malloc(size);

        // Our 1D (N) variables
        size = N*sizeof(float);
        h_B = (float*)malloc(size);
        h_x = (float*)malloc(size);
        h_R = (float*)malloc(size);
        h_P = (float*)malloc(size);
        h_AP = (float*)malloc(size);

        // Small array holding scalars
        size = 5*sizeof(float);
        h_scalars = (float*)malloc(size);

	// GPU variables
	size = N*N*sizeof(float);
	hipError_t Error;
	d_A = (float*)malloc(size);
	Error = hipMalloc((void**)&d_A, size);
	if (Error != hipSuccess)
	printf("CUDA error(malloc d_A) = %s\n",hipGetErrorString(Error));

	size = N*sizeof(float);
	d_B = (float*)malloc(size);
        Error = hipMalloc((void**)&d_B, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_B) = %s\n",hipGetErrorString(Error));
	d_x = (float*)malloc(size);
        Error = hipMalloc((void**)&d_x, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_x) = %s\n",hipGetErrorString(Error));
	d_R = (float*)malloc(size);
        Error = hipMalloc((void**)&d_R, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_R) = %s\n",hipGetErrorString(Error));
	d_P =  (float*)malloc(size);
        Error = hipMalloc((void**)&d_P, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_P) = %s\n",hipGetErrorString(Error));
	d_AP =  (float*)malloc(size);
        Error = hipMalloc((void**)&d_AP, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_AP) = %s\n",hipGetErrorString(Error));

	size = 5*sizeof(float);
	d_scalars = (float*)malloc(size);
	Error = hipMalloc((void**)&d_scalars, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_scalars) = %s\n",hipGetErrorString(Error));

	size = (N+256-1)/256*sizeof(float);
        d_temp = (float*)malloc(size);
        Error = hipMalloc((void**)&d_temp, size);
        if (Error != hipSuccess)
	printf("CUDA error(malloc d_temp) = %s\n", hipGetErrorString(Error));
}


void Free_Memory() {

        // Now we better free the memory on the GPU
        free(h_A);
        free(h_B);
        free(h_x);
        free(h_R);
        free(h_P);
        free(h_AP);
        free(h_scalars);

	// Free GPU memory
	hipFree(d_A);hipFree(d_B);hipFree(d_x);
	hipFree(d_R);hipFree(d_P);hipFree(d_AP);
	hipFree(d_scalars);hipFree(d_temp);
}
/*
void Init() {

        int i;
        int index;
        int x_cell, y_cell;
        // Set up our A and B matrix

        // Full 2D Heat Transfer test
        for (i = 0; i < N; i++) {

                y_cell = (int)(i/NX);
                x_cell = i - y_cell*NX;

                // Find the diagonal
                index = i*N + i;
                h_A[index] = 2.0*PHI_X + 2.0*PHI_Y;

                // Modify A for left and right
                if (x_cell > 0) {
                        h_A[index-1] = -PHI_X;
                }

                if (x_cell < (NX-1)) {
                        h_A[index+1] = -PHI_X;
                }

                // Modifiy for up and down
                if (y_cell > 0) {
                        h_A[index-NX] = -PHI_Y;
                }
                if (y_cell < (NY-1)) {
                        h_A[index+NX] = -PHI_Y;
                }


                // Set B now
                if (y_cell == 0) {
                        h_B[i] = PHI_Y;
                } else {
                        h_B[i] = 0.0;
                }

                // And our initial guess x
                h_x[i] = 0.0;
        }


}
*/
void Read_File(){
	start_cu = clock();

	FILE *pFile;
        int i;
	// Read the matrix B
        pFile = fopen("B.txt","r");
        for (i = 0; i < N; i++) {
		h_B[i] = 0.0;
		fscanf(pFile, "%g", &h_B[i]);
	}
	fclose(pFile);

	// Rean the matrix A
	pFile = fopen("A.txt","r");
	for (i = 0; i < N*N; i++) {
                h_A[i] = 0.0;
                fscanf(pFile, "%g,", &h_A[i]);
        }
	fclose(pFile);

	// And our initial guess x
        for (i = 0; i < N; i++) {
		h_x[i] = 0.0;
	}

}


void Save_Result() {

        FILE *pFile;
        int i;//,j;
        //int index;
	/*
        pFile = fopen("A_ans.txt","w");
        // Save the matrix A
        for (i = 0; i < N; i++) {
                for (j = 0; j < N; j++) {
                        index = i*N + j;
                        fprintf(pFile, "%g", h_A[index]);
                        if (j == (N-1)) {
                                fprintf(pFile, "\n");
                        } else {
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);

        pFile = fopen("B_ans.txt","w");
        // Save the vector B
        for (i = 0; i < N; i++) {
                fprintf(pFile, "%g\n", h_B[i]);
        }
        fclose(pFile);
	*/

        pFile = fopen("X_CG.txt","w");
        // Save the vector X
        for (i = 0; i < N; i++) {
                fprintf(pFile, "%g\n", h_x[i]);
        }
        fclose(pFile);

        pFile = fopen("R_CG.txt","w");
        // Save the vector R
        for (i = 0; i < N; i++) {
                fprintf(pFile, "%g\n", h_R[i]);
        }
        fclose(pFile);


}

void Send_To_Device(){
	//cutCreateTimer(&gpu_timer);
	//hipDeviceSynchronize();
	//cutStartTimer(gpu_timer);

	hipError_t Error;
	size_t size = N*N*sizeof(float);
	Error = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	if (Error != hipSuccess)
	printf("CUDA error(copy h_A->d_A) = %s\n",hipGetErrorString(Error));
	size = N*sizeof(float);
	Error = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	if (Error != hipSuccess)
	printf("CUDA error(copy h_B->d_B) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
	printf("CUDA error(copy h_x->d_x) = %s\n",hipGetErrorString(Error));
}

__global__ void GPU_Setup(float *x, float *y, float *z, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i<n){
 		y[i] = x[i];
		z[i] = x[i];
	}
}

void SetUp_CG_GPU(){
	int tpb = 256;
	int bpg = (N+tpb-1)/tpb;

	GPU_Setup<<<bpg, tpb>>>(d_B, d_R, d_P, N);
}

__global__ void GPU_DotProduct(float *x, float *y, float *z, int n){
	int I = threadIdx.x;
	int i = blockDim.x * blockIdx.x + I;

	__shared__ float temp[256];
	temp[I] = 0.0;
	if(i<n){
		temp[I] = x[i] * y[i];
	}

	__syncthreads();

	for (int stride = blockDim.x/2; stride>0; stride = stride/2){
		if(I<stride){
			temp[I] = temp[I] + temp[I+stride];
		}
		__syncthreads();
	}

	if(I==0){
        	z[blockIdx.x] = temp[0];
        }

}

__global__ void GPU_DotProduct_Sum(float *x, float *y, int y_location, int n){

        int i;
        float sum = 0.0;
        for(i=0; i<n; i++){
                sum += x[i];
        }
        y[y_location] = sum;
}
/*
__global__ void GPU_DotProduct_Sum_reduction(float *x, float *y, int y_location){
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        for (int stride = blockDim.x/2; stride>0; stride = stride/2){
                if(i<stride){
                        x[i] = x[i] + x[i+stride];
                }
                __syncthreads();
        }

        if(i==0){
                y[y_location] = x[0];
        }
}
*/
void DotFunction1(){
	int tpb = 256;
	int bpg = (N+tpb-1)/tpb;

	GPU_DotProduct<<<bpg, tpb>>>(d_P, d_AP, d_temp, N);
	GPU_DotProduct_Sum<<<1, 1>>>(d_temp, d_scalars, 2, bpg);
	//GPU_DotProduct_Sum_reduction<<<1, 4>>>(d_temp, d_scalars, 2);
	GPU_DotProduct<<<bpg, tpb>>>(d_R, d_R, d_temp, N);
	GPU_DotProduct_Sum<<<1, 1>>>(d_temp, d_scalars, 3, bpg);
	//GPU_DotProduct_Sum_reduction<<<1, 4>>>(d_temp, d_scalars, 3);
}

void DotFunction2(){
        int tpb = 256;
        int bpg = (N+tpb-1)/tpb;

        GPU_DotProduct<<<bpg, tpb>>>(d_R, d_R, d_temp, N);
	GPU_DotProduct_Sum<<<1, 1>>>(d_temp, d_scalars, 4, bpg);
	//GPU_DotProduct_Sum_reduction<<<1, 4>>>(d_temp, d_scalars, 4);
}


__global__ void GPU_ScalarsCal_alpha(float *x){
	x[0] = x[3]/x[2];
}

void Cal_alpha(){
	int tpb = 1;
        int bpg = 1;

	GPU_ScalarsCal_alpha<<<bpg, tpb>>>(d_scalars);
}

__global__ void GPU_ScalarsCal_beta(float *x){
        x[1] = x[4]/x[3];
}

void Cal_beta(){
        int tpb = 1;
        int bpg = 1;

        GPU_ScalarsCal_beta<<<bpg, tpb>>>(d_scalars);
}

__global__ void GPU_MatrixVectorProduct(float *x, float *y, float *z, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j;

	if(i<n){
		z[i] = 0.0;
		for(j=0;j<n;j++){
			z[i] = z[i] + x[i*n+j]*y[j];
		}
		
	}
} 

void MatrixVectorFunction(){
	int tpb = 256;
	int bpg = (N+tpb-1)/tpb;

	GPU_MatrixVectorProduct<<<bpg, tpb>>>(d_A, d_P, d_AP, N);
}

__global__ void GPU_Update_x(float *x, float *y, float *z, int n){
	int j = threadIdx.x;
        int i = blockDim.x * blockIdx.x + j;

        __shared__ float sx[256];
        __shared__ float sy[256];

        sx[j] = x[i];
        sy[j] = y[i];

        if(i<n) sx[j] = sx[j] + z[0] * sy[j];

        x[i] = sx[j];

	//if(i<n) x[i] = x[i] + z[0] * y[i];
}

__global__ void GPU_Update_R(float *x, float *y, float *z, int n){
        int j = threadIdx.x;
        int i = blockDim.x * blockIdx.x + j;

        __shared__ float sx[256];
        __shared__ float sy[256];

        sx[j] = x[i];
        sy[j] = y[i];

        if(i<n) sx[j] = sx[j] - z[0] * sy[j];

        x[i] = sx[j];

	//if(i<n) x[i] = x[i] - z[0] * y[i];
}

void Update_xandR(){
	int tpb = 256;
        int bpg = (N+tpb-1)/tpb;

	GPU_Update_x<<<bpg, tpb>>>(d_x, d_P, d_scalars, N);
	GPU_Update_R<<<bpg, tpb>>>(d_R, d_AP, d_scalars, N);
}

__global__ void GPU_Update_P(float *x, float *y, float *z, int n){
	int j = threadIdx.x;
        int i = blockDim.x * blockIdx.x + j;

        __shared__ float sx[256];
        __shared__ float sy[256];

        sx[j] = x[i];
        sy[j] = y[i];

        if(i<n) sx[j] = sy[j] + z[1] * sx[j];

        x[i] = sx[j];

	//if(i<n) x[i] = y[i] + z[1] * x[i];
}

void Update_P(){
	int tpb = 256;
        int bpg = (N+tpb-1)/tpb;

        GPU_Update_P<<<bpg, tpb>>>(d_P, d_R, d_scalars, N);
}

void Send_For_Print(){
	hipError_t Error;
        size_t size = 5*sizeof(float);
        Error = hipMemcpy(h_scalars, d_scalars, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_scalars->h_scalars) = %s\n",hipGetErrorString(Error));
}

void Send_To_Host(){
	hipError_t Error;
        size_t size = N*sizeof(float);
	/*
        Error = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_B->h_B) = %s\n",hipGetErrorString(Error));
	*/
	Error = hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_x->h_x) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_R->h_R) = %s\n",hipGetErrorString(Error));
	/*
	size = N*N*sizeof(float);
	Error = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_A->h_A) = %s\n",hipGetErrorString(Error));
	*/

	end_cu = clock();
        time_used_cu = (float)(end_cu - start_cu)/ CLOCKS_PER_SEC;
        printf("\ntime in cu = %f s\n",time_used_cu);
	//hipDeviceSynchronize();
}
