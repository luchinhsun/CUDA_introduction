#include "head.h"

float *h_a;
float *h_b;

float *d_a;

void CPU_malloc(){

	size_t size = N*sizeof(float);

	h_a = (float *)malloc(size);
	h_b = (float *)malloc(size);
}

void GPU_malloc(){

	size_t size = N*sizeof(float);

	hipError_t Error;

	Error = hipMalloc((void**)&d_a,size);
	printf("CUDA error(malloc d_a) = %s\n", hipGetErrorString(Error));
}

void Free(){

	free(h_a);
	free(h_b);

	hipFree(d_a);
}
